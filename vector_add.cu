
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

__global__ void parallel_vector_add(int* d_a, int* d_b, int* d_c, int* d_n) {
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(i < *d_n) {
        printf("I am about to compute c[%d].\n", i);
        d_c[i] = d_a[i] + d_b[i];
    }else {
        printf("I am therad #%d, and doing nothing.\n", i);
    }
}

int main() {
    int n;

    scanf("%d", &n);

    int h_a[n];
    int h_b[n];
    for(int i=0; i<n; i++) {
        h_a[i] = i;
        h_b[i] = n-i;
    }

    int h_c[n];

    int* d_a, *d_b, *d_c, *d_n;
    hipMalloc((void **) &d_a, n*sizeof(int));
    hipMalloc((void **) &d_b, n*sizeof(int));
    hipMalloc((void **) &d_c, n*sizeof(int));
    hipMalloc((void **) &d_n, sizeof(int));

    hipMemcpy(d_a, &h_a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &h_b, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);

    int block = ceil(n/10.0);

    parallel_vector_add<<<block, 10>>>(d_a, d_b, d_c, d_n);
    hipDeviceSynchronize();

    hipMemcpy(&h_c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    for(int i = 0; i < n; i++)
        printf("%d ", h_c[i]);
}